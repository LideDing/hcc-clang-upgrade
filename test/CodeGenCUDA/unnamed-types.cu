#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -std=c++11 -x hip -triple x86_64-linux-gnu -fcuda-force-lambda-odr -emit-llvm %s -o - | FileCheck %s --check-prefix=HOST
// RUN: %clang_cc1 -std=c++11 -x hip -triple amdgcn-amd-amdhsa -fcuda-force-lambda-odr -fcuda-is-device -emit-llvm %s -o - | FileCheck %s --check-prefix=DEVICE

#include "Inputs/hip/hip_runtime.h"

// HOST: @0 = private unnamed_addr constant [43 x i8] c"_Z2k0IZZ2f1PfENKUlS0_E_clES0_EUlfE_EvS0_T_\00", align 1

__device__ float d0(float x) {
  return [](float x) { return x + 2.f; }(x);
}

__device__ float d1(float x) {
  return [](float x) { return x * 2.f; }(x);
}

// DEVICE: amdgpu_kernel void @_Z2k0IZZ2f1PfENKUlS0_E_clES0_EUlfE_EvS0_T_(
template <typename F>
__global__ void k0(float *p, F f) {
  p[0] = f(p[0]) + d0(p[1]) + d1(p[2]);
}

void f0(float *p) {
  [](float *p) {
    *p = 1.f;
  }(p);
}

void f1(float *p) {
  [](float *p) {
    k0<<<1,1>>>(p, [] __device__ (float x) { return x + 1.f; });
  }(p);
}
// HOST: @__hip_register_globals
// HOST: __hipRegisterFunction{{.*}}@{{(__device_stub_)?}}_Z2k0IZZ2f1PfENKUlS0_E_clES0_EUlfE_EvS0_T_{{.*}}@0
